// multi_gpu_miner.cu

extern "C" {
#include <stdint.h>
}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <thread>
#include <vector>
#include <atomic>

#define HASH_SIZE 32
#define INPUT_SIZE 40
#define BLOCK_SIZE 1024
#define GRID_SIZE 65535

__device__ void keccak256(const uint8_t *input, size_t input_len, uint8_t *output);

__device__ bool is_hash_less(const uint8_t *hash, const uint8_t *max_value) {
    for (int i = 0; i < HASH_SIZE; ++i) {
        if (hash[i] < max_value[i]) return true;
        if (hash[i] > max_value[i]) return false;
    }
    return false;
}

__global__ void keccak_kernel(
    const uint8_t *prev_hash,
    const uint8_t *max_value,
    uint64_t start_nonce,
    uint64_t *found_nonce,
    int *found_flag
) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t nonce = start_nonce + idx;

    if (*found_flag) return;

    uint8_t input[INPUT_SIZE];
    uint8_t hash[HASH_SIZE];

    for (int i = 0; i < 32; ++i)
        input[i] = prev_hash[i];

    for (int i = 0; i < 8; ++i)
        input[32 + i] = (nonce >> ((7 - i) * 8)) & 0xff;

    keccak256(input, INPUT_SIZE, hash);

    if (is_hash_less(hash, max_value)) {
        if (atomicCAS(found_flag, 0, 1) == 0) {
            *found_nonce = nonce;
        }
    }
}

extern "C" void keccak_miner_multi_gpu(
    uint8_t *prev_hash,
    uint8_t *max_value,
    uint64_t start_nonce,
    uint64_t *found_nonce,
    int *found_flag
) {
    int device_count = 0;
    hipGetDeviceCount(&device_count);

    std::atomic<bool> found(false);
    std::vector<std::thread> threads;

    for (int dev = 0; dev < device_count; ++dev) {
        threads.emplace_back([=, &found]() {
            hipSetDevice(dev);

            uint8_t *d_prev, *d_max;
            uint64_t *d_found_nonce;
            int *d_found_flag;

            hipMalloc(&d_prev, 32);
            hipMalloc(&d_max, 32);
            hipMalloc(&d_found_nonce, sizeof(uint64_t));
            hipMalloc(&d_found_flag, sizeof(int));

            hipMemcpy(d_prev, prev_hash, 32, hipMemcpyHostToDevice);
            hipMemcpy(d_max, max_value, 32, hipMemcpyHostToDevice);
            hipMemcpy(d_found_nonce, found_nonce, sizeof(uint64_t), hipMemcpyHostToDevice);
            hipMemcpy(d_found_flag, found_flag, sizeof(int), hipMemcpyHostToDevice);

            uint64_t offset_nonce = start_nonce + (uint64_t)dev * BLOCK_SIZE * GRID_SIZE;

            keccak_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(
                d_prev, d_max, offset_nonce, d_found_nonce, d_found_flag
            );

            hipDeviceSynchronize();

            int host_flag = 0;
            hipMemcpy(&host_flag, d_found_flag, sizeof(int), hipMemcpyDeviceToHost);
            if (host_flag == 1 && !found.exchange(true)) {
                hipMemcpy(found_nonce, d_found_nonce, sizeof(uint64_t), hipMemcpyDeviceToHost);
                *found_flag = 1;
            }

            hipFree(d_prev);
            hipFree(d_max);
            hipFree(d_found_nonce);
            hipFree(d_found_flag);
        });
    }

    for (auto &t : threads) {
        t.join();
    }
}
